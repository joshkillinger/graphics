#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//#include <helper_functions.h>
#ifdef _WIN32
#include <windows.h>
#else
#include <sys/time.h>
#endif
#include <iomanip>

using namespace std;

/*
 *  Return elapsed wall time since last call (seconds)
 */
static double t0=0;
double Elapsed(void)
{
#ifdef _WIN32
   //  Windows version of wall time
   LARGE_INTEGER tv,freq;
   QueryPerformanceCounter((LARGE_INTEGER*)&tv);
   QueryPerformanceFrequency((LARGE_INTEGER*)&freq);
   double t = tv.QuadPart/(double)freq.QuadPart;
#else
   //  Unix/Linux/OSX version of wall time
   struct timeval tv;
   gettimeofday(&tv,NULL);
   double t = tv.tv_sec+1e-6*tv.tv_usec;
#endif
   double s = t-t0;
   t0 = t;
   return s;
}

/*
 *  Initialize matrix with random values
 */
void RandomInit(float x[],const unsigned int n)
{
    for (unsigned int i=0;i<n;i++)
        x[i] = rand() / (float)RAND_MAX;
}

/*
 *  Initialize fastest GPU device
 */
int InitGPU(int verbose)
{
    //  Get number of CUDA devices
    int num;
    if (hipGetDeviceCount(&num))
    {
        cerr << "Cannot get number of CUDA devices" << endl;
        exit(1);
    }
    if (num<1)
    {
        cerr << "No CUDA devices found\n" << endl;
        exit(1);
    }

    //  Get fastest device
    hipDeviceProp_t prop;
    int   MaxDevice = -1;
    int   MaxGflops = -1;
    for (int dev=0;dev<num;dev++)
    {
        if (hipGetDeviceProperties(&prop,dev))
        {
           cerr << "Error getting device " << dev << " properties" << endl;
           exit(1);
        }
        int Gflops = prop.multiProcessorCount * prop.clockRate;
        if (verbose)
        {
            cout << "CUDA Device " << dev << ": " << prop.name << " Gflops " << (1e-6*Gflops) << " Processors " << prop.multiProcessorCount << " Threads/Block " << prop.maxThreadsPerBlock << endl;
        }
        if(Gflops > MaxGflops)
        {
            MaxGflops = Gflops;
            MaxDevice = dev;
        }
    }

    //  Print and set device
    if (hipGetDeviceProperties(&prop,MaxDevice))
    {
        cerr << "Error getting device " << MaxDevice << " properties" << endl;
    }
    cout << "Fastest CUDA Device " << MaxDevice << ": " << prop.name << endl;
    hipSetDevice(MaxDevice);

    //  Return max thread count
    return prop.maxThreadsPerBlock;
}

/*
 * c = a dot B -- host
 */
void Doth(float c[], const float a[], const float B[], unsigned int n)
{
    for (unsigned int i = 0; i < n; i++)
    {
        double sum = 0;
        for (unsigned int j = 0; j < n; j++)
        {
            sum += (double)a[j] * (double)B[(i * n) + j];
        }
        c[i] = (float)sum;
    }
}

/*
 * Compute c[i] = a dot B[i]
 */
__global__ void Dot(float c[], const float a[], const float B[], const unsigned int n, const unsigned totalThreads)
{
    int vectorsPerThread = n / totalThreads;
    unsigned int i = (blockIdx.x*blockDim.x+threadIdx.x) * vectorsPerThread;
    for (int v = 0; v < vectorsPerThread; v++)
    {
        float sum = 0;
        for (int j = 0; j < n; j++)
        {
            sum += a[j] * B[((i + v) * n) + j];
        }
        c[i + v] = sum;
    }
}

void Dotd(float ch[],const float ah[],const float Bh[],const unsigned int n)
{
    //  Calculate matrix dimensions
    int N = n*n*sizeof(float);

    // Allocate device memory
    float* ad;
    float* Bd;
    float* cd;
    if (hipMalloc((void**)&ad, n * sizeof(float)))
    {
        cerr << "Cannot allocate device memory ad" << endl;
        exit(1);
    }
    if (hipMalloc((void**)&Bd, N))
    {
        cerr << "Cannot allocate device memory Bd" << endl;
        exit(1);
    }
    if (hipMalloc((void**)&cd, n * sizeof(float)))
    {
        cerr << "Cannot allocate device memory cd" << endl;
        exit(1);
    }
    // Copy A and B from host to device
    if (hipMemcpy(ad, ah, n * sizeof(float), hipMemcpyHostToDevice))
    {
        cerr << "Cannot copy a from host to device" << endl;
        exit(1);
    }
    if (hipMemcpy(Bd, Bh, N, hipMemcpyHostToDevice))
    {
        cerr << "Cannot copy B from host to device" << endl;
        exit(1);
    }

    // Set size of block to 32, and 32 blocks
    int totalthreads = 32*32;
    dim3 threads(32);
    dim3 grid(32);
    // Execute the kernel
    Dot<<<grid,threads>>>(cd, ad, Bd, n, totalthreads);
    if (hipGetLastError())
    {
        cerr << "Dot failed" << endl;
        exit(1);
    }

    // Copy C from device to host
    if (hipMemcpy(ch, cd, n * sizeof(float), hipMemcpyDeviceToHost))
    {
        cerr << "Cannot copy C from device to host" << endl;
        exit(1);
    }

    //  Free device memory
    hipFree(ad);
    hipFree(Bd);
    hipFree(cd);
}

void DotMul(int maxThreadsPerBlock)
{
    int n = 1024 * 10;
    int N = n * n * sizeof(float);

    cout << n << " vectors of " << n << " length = " << N << " bytes" << endl;

    // Allocate host matrices A/B/C/R
    float* ah = (float*)malloc(n * sizeof(float));
    float* Bh = (float*)malloc(N);
    float* ch = (float*)malloc(n * sizeof(float));
    float* rh = (float*)malloc(n * sizeof(float));
    if (!ah || !Bh || !ch || !rh)
    {
        cerr << "Cannot allocate host memory" << endl;
        exit(1);
    }

    // Initialize a & B
    srand(9999);
    RandomInit(ah,n);
    RandomInit(Bh,n*n);

    //  Compute r = a dot B on host
    Elapsed();
    Doth(rh,ah,Bh,n);
    double Th = Elapsed();

    //  Compute c = a dot B on device
    Elapsed();
    Dotd(ch,ah,Bh,n);
    double Td = Elapsed();

    //  Compute difference between R and C
    double r2=0;
    for (int i = 0; i < n; i++)
        r2 += fabs(ch[i]-rh[i]);
    r2 /= n;

    cout << "Host results: ";
/*    for (int i = 0; i < n; i++)
    {
        cout << rh[i] << ", ";
    }
*/
    cout << rh[0] << ", " << rh[1] << ", ..., " << rh[n-2] << ", " << rh[n-1];

    cout << endl << "Device results: ";
/*    for (int i = 0; i < n; i++)
    {
        cout << ch[i] << ", ";
    }
*/
    cout << ch[0] << ", " << ch[1] << ", ..., " << ch[n-2] << ", " << ch[n-1];
    cout << endl;

    //  Free host memory
    free(ah);
    free(Bh);
    free(ch);
    free(rh);

    //  Print results
    cout << "Host   Time = " << Th << " s" << endl;
    cout << "Device Time = " << Td << " s" << endl;
    cout << "Speedup = " << Th/Td << endl;
    cout << "Difference = " << r2 << endl;
}

int main(int argc, char** argv)
{
    cout << "Josh Killinger: HW10" << endl;
    cout << setw(8) << setprecision(4);

	int maxThreadsPerBlock = InitGPU(1);

    DotMul(maxThreadsPerBlock);

#ifdef _WIN32
    char c;
    cin >> c;
#endif

    return 0;
}
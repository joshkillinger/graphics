#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//#include <helper_functions.h>
#ifdef _WIN32
#include <windows.h>
#else
#include <sys/time.h>
#endif
#include <iomanip>

using namespace std;

/*
 *  Return elapsed wall time since last call (seconds)
 */
static double t0=0;
double Elapsed(void)
{
#ifdef _WIN32
   //  Windows version of wall time
   LARGE_INTEGER tv,freq;
   QueryPerformanceCounter((LARGE_INTEGER*)&tv);
   QueryPerformanceFrequency((LARGE_INTEGER*)&freq);
   double t = tv.QuadPart/(double)freq.QuadPart;
#else
   //  Unix/Linux/OSX version of wall time
   struct timeval tv;
   gettimeofday(&tv,NULL);
   double t = tv.tv_sec+1e-6*tv.tv_usec;
#endif
   double s = t-t0;
   t0 = t;
   return s;
}

/*
 *  Initialize matrix with random values
 */
void RandomInit(float x[],const unsigned int n)
{
    for (unsigned int i=0;i<n;i++)
        x[i] = rand() / (float)RAND_MAX;
}

/*
 *  Initialize fastest GPU device
 */
int InitGPU(int verbose)
{
    //  Get number of CUDA devices
    int num;
    if (hipGetDeviceCount(&num))
    {
        cerr << "Cannot get number of CUDA devices" << endl;
        exit(1);
    }
    if (num<1)
    {
        cerr << "No CUDA devices found\n" << endl;
        exit(1);
    }

    //  Get fastest device
    hipDeviceProp_t prop;
    int   MaxDevice = -1;
    int   MaxGflops = -1;
    for (int dev=0;dev<num;dev++)
    {
        if (hipGetDeviceProperties(&prop,dev))
        {
           cerr << "Error getting device " << dev << " properties" << endl;
           exit(1);
        }
        int Gflops = prop.multiProcessorCount * prop.clockRate;
        if (verbose)
        {
            cout << "CUDA Device " << dev << ": " << prop.name << " Gflops " << (1e-6*Gflops) << " Processors " << prop.multiProcessorCount << " Threads/Block " << prop.maxThreadsPerBlock << endl;
        }
        if(Gflops > MaxGflops)
        {
            MaxGflops = Gflops;
            MaxDevice = dev;
        }
    }

    //  Print and set device
    if (hipGetDeviceProperties(&prop,MaxDevice))
    {
        cerr << "Error getting device " << MaxDevice << " properties" << endl;
    }
    cout << "Fastest CUDA Device " << MaxDevice << ": " << prop.name << endl;
    hipSetDevice(MaxDevice);

    //  Return max thread count
    return prop.maxThreadsPerBlock;
}

/*
 * C = A * B -- host
 */
void AxBh(float C[], const float A[], const float B[], unsigned int n)
{
   for (unsigned int i=0;i<n;i++)
      for (unsigned int j=0;j<n;j++)
      {
         double sum=0;
         for (unsigned int k=0;k<n;k++)
            sum += (double)A[i*n+k] * (double)B[k*n+j];
         C[i*n+j] = (float)sum;
      }
}



/*
 * Compute one element of A * B
 */
__global__ void AxB(float C[],const float A[],const float B[],const unsigned int n)
{
   unsigned int j = blockIdx.x*blockDim.x+threadIdx.x;
   unsigned int i = blockIdx.y*blockDim.y+threadIdx.y;
   float sum =0;
   for (int k=0;k<n;k++)
      sum += A[i*n+k] * B[k*n+j];
   C[i*n+j] = sum;
}

/*
 * C = A * B -- device
 */
void AxBd(float Ch[],const float Ah[],const float Bh[],const unsigned int Bw,const unsigned int Bn)
{
    //  Calculate matrix dimensions
    int n = Bw*Bn;
    int N = n*n*sizeof(float);

    // Allocate device memory
    float* Ad;
    float* Bd;
    float* Cd;
    if (hipMalloc((void**)&Ad,N))
    {
        cerr << "Cannot allocate device memory Ad" << endl;
        exit(1);
    }
    if (hipMalloc((void**)&Bd,N))
    {
        cerr << "Cannot allocate device memory Bd" << endl;
        exit(1);
    }
    if (hipMalloc((void**)&Cd,N))
    {
        cerr << "Cannot allocate device memory Cd" << endl;
        exit(1);
    }
    // Copy A and B from host to device
    if (hipMemcpy(Ad,Ah,N,hipMemcpyHostToDevice))
    {
        cerr << "Cannot copy A from host to device" << endl;
        exit(1);
    }
    if (hipMemcpy(Bd,Bh,N,hipMemcpyHostToDevice))
    {
        cerr << "Cannot copy B from host to device" << endl;
        exit(1);
    }

    // Set size of block to Bw x Bw, and Bn x Bn blocks
    dim3 threads(Bw,Bw);
    dim3 grid(Bn,Bn);
    // Execute the kernel
    AxB<<<grid,threads>>>(Cd,Ad,Bd,n);
    if (hipGetLastError())
    {
        cerr << "AxB failed" << endl;
        exit(1);
    }

    // Copy C from device to host
    if (hipMemcpy(Ch,Cd,N,hipMemcpyDeviceToHost))
    {
        cerr << "Cannot copy C from device to host" << endl;
        exit(1);
    }

    //  Free device memory
    hipFree(Ad);
    hipFree(Bd);
    hipFree(Cd);
}

void MatMul(int maxThreadsPerBlock)
{
    //  Total width is block times number of blocks
    int Bw = 32;
    int Bn = 32;
    int n = Bw*Bn;
    int N = n*n*sizeof(float);
    cout << "Bw=" << Bw << " Bn=" << Bn << " n=" << n << endl;

    // Allocate host matrices A/B/C/R
    float* Ah = (float*)malloc(N);
    float* Bh = (float*)malloc(N);
    float* Ch = (float*)malloc(N);
    float* Rh = (float*)malloc(N);
    if (!Ah || !Bh || !Ch || !Rh)
    {
        cerr << "Cannot allocate host memory" << endl;
        exit(1);
    }

    // Initialize A & B
    srand(9999);
    RandomInit(Ah,n*n);
    RandomInit(Bh,n*n);

    //  Compute R = AB on host
    Elapsed();
    AxBh(Rh,Ah,Bh,n);
    double Th = Elapsed();

    //  Compute C = AB on device
    Elapsed();
    AxBd(Ch,Ah,Bh,Bw,Bn);
    double Td = Elapsed();

    //  Compute difference between R and C
    double r2=0;
    for (int i=0;i<n*n;i++)
        r2 += fabs(Ch[i]-Rh[i]);
    r2 /= n*n;

    //  Free host memory
    free(Ah);
    free(Bh);
    free(Ch);
    free(Rh);

    //  Print results
    cout << "Host   Time = " << Th << " s" << endl;
    cout << "Device Time = " << Td << " s" << endl;
    cout << "Speedup = " << Th/Td << endl;
    cout << "Difference = " << r2 << endl;
}

/*
 * c = a dot B -- host
 */
void Doth(float c[], const float a[], const float B[], unsigned int n)
{
    for (unsigned int i = 0; i < n; i++)
    {
        //double sum = 0;
        //for (unsigned int j = 0; j < n; j++)
        //{
        //    sum += (double)a[j] * (double)B[(i * n) + j];
        //}
        //c[i] = (float)sum;
        c[i] = i;
    }
}

/*
 * Compute a dot B[i]
 */
__global__ void Dot(float c[], const float a[], const float B[], const unsigned int n)
{
   unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
   //float sum =0;
   //for (int j = 0; j < n; j++)
   //   sum += a[j] * B[(i*n)+j];
   //c[i] = sum;
   c[i] = i;
}

void Dotd(float ch[],const float ah[],const float Bh[],const unsigned int n)
{
    //  Calculate matrix dimensions
    int N = n*n*sizeof(float);

    // Allocate device memory
    float* ad;
    float* Bd;
    float* cd;
    if (hipMalloc((void**)&ad, n * sizeof(float)))
    {
        cerr << "Cannot allocate device memory Ad" << endl;
        exit(1);
    }
    if (hipMalloc((void**)&Bd, N))
    {
        cerr << "Cannot allocate device memory Bd" << endl;
        exit(1);
    }
    if (hipMalloc((void**)&cd, n * sizeof(float)))
    {
        cerr << "Cannot allocate device memory Cd" << endl;
        exit(1);
    }
    // Copy A and B from host to device
    if (hipMemcpy(ad, ah, n * sizeof(float), hipMemcpyHostToDevice))
    {
        cerr << "Cannot copy A from host to device" << endl;
        exit(1);
    }
    if (hipMemcpy(Bd, Bh, N, hipMemcpyHostToDevice))
    {
        cerr << "Cannot copy B from host to device" << endl;
        exit(1);
    }

    // Set size of block to 32, and 32 blocks
    dim3 threads(32);
    dim3 grid(32);
    // Execute the kernel
    AxB<<<grid,threads>>>(cd, ad, Bd, n);
    if (hipGetLastError())
    {
        cerr << "Dot failed" << endl;
        exit(1);
    }

    // Copy C from device to host
    if (hipMemcpy(ch, cd, n * sizeof(float), hipMemcpyDeviceToHost))
    {
        cerr << "Cannot copy C from device to host" << endl;
        exit(1);
    }

    //  Free device memory
    hipFree(ad);
    hipFree(Bd);
    hipFree(cd);
}

void MegaDot(int maxThreadsPerBlock)
{
    int n = 1024;
    int N = n * n * sizeof(float);

    cout << n << " vectors of " << n << " length = " << N << " bytes" << endl;

    // Allocate host matrices A/B/C/R
    float* ah = (float*)malloc(n * sizeof(float));
    float* Bh = (float*)malloc(N);
    float* ch = (float*)malloc(n * sizeof(float));
    float* rh = (float*)malloc(n * sizeof(float));
    if (!ah || !Bh || !ch || !rh)
    {
        cerr << "Cannot allocate host memory" << endl;
        exit(1);
    }

    // Initialize a & B
    srand(9999);
    RandomInit(ah,n);
    RandomInit(Bh,n*n);

    //  Compute r = a dot B on host
    Elapsed();
    Doth(rh,ah,Bh,n);
    double Th = Elapsed();

    //  Compute c = a dot B on device
    Elapsed();
    Dotd(ch,ah,Bh,n);
    double Td = Elapsed();

    //  Compute difference between R and C
    double r2=0;
    for (int i = 0; i < n; i++)
        r2 += fabs(ch[i]-rh[i]);
    r2 /= n;

    cout << "Host results: ";
/*    for (int i = 0; i < n; i++)
    {
        cout << rh[i] << ", ";
    }
*/
    cout << rh[0] << ", " << rh[1] << ", ..., " << rh[n-2] << ", " << rh[n-1];

    cout << endl << "Device results: ";
/*    for (int i = 0; i < n; i++)
    {
        cout << ch[i] << ", ";
    }
*/
    cout << ch[0] << ", " << ch[1] << ", ..., " << ch[n-2] << ", " << ch[n-1];
    cout << endl;

    //  Free host memory
    free(ah);
    free(Bh);
    free(ch);
    free(rh);

    //  Print results
    cout << "Host   Time = " << Th << " s" << endl;
    cout << "Device Time = " << Td << " s" << endl;
    cout << "Speedup = " << Th/Td << endl;
    cout << "Difference = " << r2 << endl;
}

int main(int argc, char** argv)
{
    cout << "Josh Killinger: HW10" << endl;
    cout << setw(7) << setprecision(4);

	int maxThreadsPerBlock = InitGPU(1);

    //MatMul(maxThreadsPerBlock);

    MegaDot(maxThreadsPerBlock);

    char c;
    cin >> c;

    return 0;
}
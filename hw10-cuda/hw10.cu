#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <windows.h>
#include <iomanip>

using namespace std;

/*
 *  Return elapsed wall time since last call (seconds)
 */
static double t0=0;
double Elapsed(void)
{
   //  Windows version of wall time
   LARGE_INTEGER tv,freq;
   QueryPerformanceCounter((LARGE_INTEGER*)&tv);
   QueryPerformanceFrequency((LARGE_INTEGER*)&freq);
   double t = tv.QuadPart/(double)freq.QuadPart;
   double s = t-t0;
   t0 = t;
   return s;
}

/*
 *  Initialize matrix with random values
 */
void RandomInit(float x[],const unsigned int n)
{
    for (unsigned int i=0;i<n*n;i++)
        x[i] = rand() / (float)RAND_MAX;
}

/*
 *  Initialize fastest GPU device
 */
int InitGPU(int verbose)
{
    //  Get number of CUDA devices
    int num;
    if (hipGetDeviceCount(&num))
    {
        cerr << "Cannot get number of CUDA devices" << endl;
        exit(1);
    }
    if (num<1)
    {
        cerr << "No CUDA devices found\n" << endl;
        exit(1);
    }

    //  Get fastest device
    hipDeviceProp_t prop;
    int   MaxDevice = -1;
    int   MaxGflops = -1;
    for (int dev=0;dev<num;dev++)
    {
        if (hipGetDeviceProperties(&prop,dev))
        {
           cerr << "Error getting device " << dev << " properties" << endl;
           exit(1);
        }
        int Gflops = prop.multiProcessorCount * prop.clockRate;
        if (verbose)
        {
            cout << "CUDA Device " << dev << ": " << prop.name << " Gflops " << (1e-6*Gflops) << " Processors " << prop.multiProcessorCount << " Threads/Block " << prop.maxThreadsPerBlock << endl;
        }
        if(Gflops > MaxGflops)
        {
            MaxGflops = Gflops;
            MaxDevice = dev;
        }
    }

    //  Print and set device
    if (hipGetDeviceProperties(&prop,MaxDevice))
    {
        cerr << "Error getting device " << MaxDevice << " properties" << endl;
    }
    cout << "Fastest CUDA Device " << MaxDevice << ": " << prop.name << endl;
    hipSetDevice(MaxDevice);

    //  Return max thread count
    return prop.maxThreadsPerBlock;
}

/*
 * C = A * B -- host
 */
void AxBh(float C[], const float A[], const float B[], unsigned int n)
{
   for (unsigned int i=0;i<n;i++)
      for (unsigned int j=0;j<n;j++)
      {
         double sum=0;
         for (unsigned int k=0;k<n;k++)
            sum += (double)A[i*n+k] * (double)B[k*n+j];
         C[i*n+j] = (float)sum;
      }
}

/*
 * Compute one element of A * B
 */
__global__ void AxB(float C[],const float A[],const float B[],const unsigned int n)
{
   unsigned int j = blockIdx.x*blockDim.x+threadIdx.x;
   unsigned int i = blockIdx.y*blockDim.y+threadIdx.y;
   float sum =0;
   for (int k=0;k<n;k++)
      sum += A[i*n+k] * B[k*n+j];
   C[i*n+j] = sum;
}

/*
 * C = A * B -- device
 */
void AxBd(float Ch[],const float Ah[],const float Bh[],const unsigned int Bw,const unsigned int Bn)
{
    //  Calculate matrix dimensions
    int n = Bw*Bn;
    int N = n*n*sizeof(float);

    // Allocate device memory
    float* Ad;
    float* Bd;
    float* Cd;
    if (hipMalloc((void**)&Ad,N))
    {
        cerr << "Cannot allocate device memory Ad" << endl;
        exit(1);
    }
    if (hipMalloc((void**)&Bd,N))
    {
        cerr << "Cannot allocate device memory Bd" << endl;
        exit(1);
    }
    if (hipMalloc((void**)&Cd,N))
    {
        cerr << "Cannot allocate device memory Cd" << endl;
        exit(1);
    }
    // Copy A and B from host to device
    if (hipMemcpy(Ad,Ah,N,hipMemcpyHostToDevice))
    {
        cerr << "Cannot copy A from host to device" << endl;
        exit(1);
    }
    if (hipMemcpy(Bd,Bh,N,hipMemcpyHostToDevice))
    {
        cerr << "Cannot copy B from host to device" << endl;
        exit(1);
    }

    // Set size of block to Bw x Bw, and Bn x Bn blocks
    dim3 threads(Bw,Bw);
    dim3 grid(Bn,Bn);
    // Execute the kernel
    AxB<<<grid,threads>>>(Cd,Ad,Bd,n);
    if (hipGetLastError())
    {
        cerr << "AxB failed" << endl;
        exit(1);
    }

    // Copy C from device to host
    if (hipMemcpy(Ch,Cd,N,hipMemcpyDeviceToHost))
    {
        cerr << "Cannot copy C from device to host" << endl;
        exit(1);
    }

    //  Free device memory
    hipFree(Ad);
    hipFree(Bd);
    hipFree(Cd);
}

int main(int argc, char** argv)
{
    cout << "Josh Killinger: HW10" << endl;

	int maxThreadsPerBlock = InitGPU(1);

    //  Total width is block times number of blocks
    int Bw = 32;
    int Bn = 32;
    int n = Bw*Bn;
    int N = n*n*sizeof(float);
    cout << "Bw=" << Bw << " Bn=" << Bn << " n=" << n << endl;

    // Allocate host matrices A/B/C/R
    float* Ah = (float*)malloc(N);
    float* Bh = (float*)malloc(N);
    float* Ch = (float*)malloc(N);
    float* Rh = (float*)malloc(N);
    if (!Ah || !Bh || !Ch || !Rh)
    {
        cerr << "Cannot allocate host memory" << endl;
        exit(1);
    }

    // Initialize A & B
    srand(9999);
    RandomInit(Ah,n);
    RandomInit(Bh,n);

    //  Compute R = AB on host
    Elapsed();
    AxBh(Rh,Ah,Bh,n);
    double Th = Elapsed();

    //  Compute C = AB on device
    Elapsed();
    AxBd(Ch,Ah,Bh,Bw,Bn);
    double Td = Elapsed();

    //  Compute difference between R and C
    double r2=0;
    for (int i=0;i<n*n;i++)
        r2 += fabs(Ch[i]-Rh[i]);
    r2 /= n*n;

    //  Free host memory
    free(Ah);
    free(Bh);
    free(Ch);
    free(Rh);


    //  Print results
    cout << setw(7) << setprecision(4);
    cout << "Host   Time = " << Th << " s" << endl;
    cout << "Device Time = " << Td << " s" << endl;
    cout << "Speedup = " << Th/Td << endl;
    cout << "Difference = " << r2 << endl;

    return 0;
}